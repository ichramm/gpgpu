#include "hip/hip_runtime.h"
/*!
 * \file ejercicio2.cu
 * \author Juan Ramirez (juan.ramirez@fing.edu.uy)
 */

#include "lab.hpp"
#include "bl_matrix.hpp"

template <typename value_type, size_t Block_Width>
__host__ __device__ void build_dense_block(uint32_t j,
                                           const uint64_t * __restrict__ bl_bitmaps,
                                           const uint32_t * __restrict__ bl_starts,
                                           const value_type * __restrict__ values,
                                           value_type *block)
{
    auto bitmap = bl_bitmaps[j];
    auto start = bl_starts[j];
    auto offset = 0u;
    for (auto i = 0u; i < Block_Width*Block_Width; ++i) {
        auto bit = 1ULL << ((Block_Width*Block_Width) - i - 1);
        if (bitmap & bit) {
            block[i] = values[start + offset];
            ++offset;
        }
    }
}

template <typename value_type, size_t Block_Width>
__host__ void serial_spmv_kernel_host(const BLMatrix<value_type, Block_Width>& mat,
                                      const value_type * __restrict__ x,
                                      value_type *y)
{
    for (auto i = 0u; i < mat.rows / Block_Width; ++i) {
        auto block_row = i * Block_Width;
        // j runs though the block columns in the current row
        for (auto j = mat.bl_row_pointers[i]; j < mat.bl_row_pointers[i + 1]; ++j) {
            value_type block[Block_Width * Block_Width] = {0};
            build_dense_block<value_type, Block_Width>(j,
                                                       mat.bl_bitmaps.data(),
                                                       mat.bl_starts.data(),
                                                       mat.values.data(),
                                                       block);

            auto block_col = mat.bl_col_indices[j] * Block_Width;
            for (auto k = 0u; k < Block_Width; ++k) {
                for (auto l = 0u; l < Block_Width; ++l) {
                    y[block_row + k] += block[k * Block_Width + l] * x[block_col + l];
                }
            }
        }
    }
}


// one block per block??
// 2D grid of size (n/Block_Width) x (m/Block_Width)
// problem: number of blocks does not match grid dimensions
// need to consider null blocks
// idea: use nested kernels -> cannot uses shared memory
// but: I kwnow how may blocks there are
// idea: use 8x8 blocks
    // idea: un bloque por fila de bloques
    // usar distintos grupos de threads para cada bloque
    // ver si no hay quilombo al sincronizar
    // buscar la forma de hacer parallel reduce


template <typename value_type, size_t Block_Width>
__global__ void par_spmv_kernel1(const typename BLMatrix<value_type, Block_Width>::DeviceStruct mat,
                                 const value_type * __restrict__ vecX,
                                 value_type *vecY)
{
    // dense block
    //value_type block[Block_Width * Block_Width];

    // one value per row
    value_type partial_sum[Block_Width];

    if (threadIdx.x == 0) {
        partial_sum[threadIdx.y & (Block_Width-1)] = 0;
    }

    __syncthreads();

    // 2d grid
    auto block_row = blockIdx.y;
    auto block_offset = blockIdx.x;

    // coordinates inside the block
    auto y = threadIdx.y;
    auto x = threadIdx.x;

    // all threads in this block will run this for loop
    for (auto j = mat.bl_row_pointers[block_row] + block_offset; j < mat.bl_row_pointers[block_row + 1]; j += gridDim.x) {
        auto block_col = mat.bl_col_indices[j] * Block_Width;


        // assume the block has Block_Width*Block_Width threads
        auto bitmap = mat.bl_bitmaps[j];
        auto start = mat.bl_starts[j];

        auto bit = 1ULL << ((Block_Width*Block_Width) - (y * Block_Width + x) - 1);
        if (bitmap & bit) {
            auto offset =
            (bitmap - (bitmap&(bit+bit-1)));
            printf("j=%d, bit=%ld, offset: %d\n", j, bit, offset);
            //block[y * Block_Width + x] = mat.values[start + offset];

            auto value = mat.values[start + offset] * vecX[block_col + x];
            //printf("pos: %d\n", y);
            //atomicAdd(&partial_sum[y], value);
            atomicAdd(&vecY[block_row * Block_Width + y], value);

        } else {
            //block[y * Block_Width + x] = 0;
        }

        __syncthreads();


        //auto value = block[y * Block_Width + x] * vecX[block_col + x];
        //printf("pos: %d\n", y);
        //atomicAdd(&partial_sum[y], value);

        __syncthreads();

        //if (threadIdx.x == 0) {
            //printf("row: %d\n", block_row * Block_Width + y);
            //atomicAdd(&vecY[block_row * Block_Width + y], partial_sum[y]);
        //}
    }
}

/////////////////////////////////
         ///  IDEA  ///
/////////////////////////////////
// Que bloque denso sea un CSR //
/////////////////////////////////

static void initial_kindergarten_test() {
    using Matrix = BLMatrix2<value_type>;

    constexpr uint32_t rows = 6;
    constexpr uint32_t cols = 6;

    std::vector<value_type> values{{1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12}};
    std::vector<uint32_t> bl_starts{{0, 2, 3, 4, 5, 7, 8, 12}};
    std::vector<uint64_t> bl_bitmaps{{9, 2, 2, 1, 3, 1, 15}};
    std::vector<uint32_t> bl_col_indices{{0, 1, 2, 0, 2, 0, 1}};
    std::vector<uint32_t> bl_row_pointers{{0, 3, 5, 7}};

    std::vector<value_type> x{{1, 1, 2, 1, 1, 2}};
    std::vector<value_type> y{{0, 0, 0, 0, 0, 0}};
    std::vector<value_type> y_expected{{1, 12, 0, 25, 28, 42}};

    Matrix mat{rows,
               cols,
               values,
               bl_starts,
               bl_bitmaps,
               bl_col_indices,
               bl_row_pointers};

    std::cout << "Matrix:" << std::endl;
    std::cout << mat << std::endl;

    serial_spmv_kernel_host<value_type, 2>(mat, x.data(), &y[0]);

    // print y
    std::cout << "y: ";
    for (auto i = 0u; i < y.size(); ++i) {
        std::cout << y[i] << " ";
    }
    std::cout << std::endl;

    auto dMat = mat.to_device();
    auto d_vecX = dev_alloc_fill(x.size(), x.data());
    auto d_vecY = dev_alloc_zero<value_type>(y.size());
    auto d_expected = dev_alloc_fill(y_expected.size(), y_expected.data());

    {
        constexpr dim3 dimGrid{1, rows/Matrix::block_width};
        constexpr dim3 dimBlock{Matrix::block_width, Matrix::block_width};
        par_spmv_kernel1<value_type, Matrix::block_width><<<dimGrid, dimBlock, Matrix::block_width>>>(dMat, d_vecX.get(), d_vecY.get());
        CUDA_CHK(hipGetLastError());
        CUDA_CHK(hipDeviceSynchronize());
        validate_results("par_spmv_kernel1", d_expected.get(), d_vecY.get(), rows);
    }

    mat.device_free(dMat);
}

void ejercicio2() {
    initial_kindergarten_test();
}
